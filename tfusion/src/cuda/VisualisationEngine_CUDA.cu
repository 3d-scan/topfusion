#include "hip/hip_runtime.h"
#include "VisualisationEngine_CUDA.h"
#include "VisualisationHelpers_CUDA.h"

using namespace tfusion;

inline dim3 getGridSize(dim3 taskSize, dim3 blockSize)
{
	return dim3((taskSize.x + blockSize.x - 1) / blockSize.x, (taskSize.y + blockSize.y - 1) / blockSize.y, (taskSize.z + blockSize.z - 1) / blockSize.z);
}

inline dim3 getGridSize(Vector2i taskSize, dim3 blockSize) { return getGridSize(dim3(taskSize.x, taskSize.y), blockSize); }

template<class TVoxel, class TIndex>
VisualisationEngine_CUDA<TVoxel, TIndex>::VisualisationEngine_CUDA(void)
{
	ORcudaSafeCall(hipMalloc((void**)&noTotalPoints_device, sizeof(uint)));
}

template<class TVoxel, class TIndex>
VisualisationEngine_CUDA<TVoxel, TIndex>::~VisualisationEngine_CUDA(void)
{
	ORcudaSafeCall(hipFree(noTotalPoints_device));
}

template<class TVoxel, class TIndex>
RenderState* VisualisationEngine_CUDA<TVoxel, TIndex>::CreateRenderState(const Scene<TVoxel, TIndex> *scene, const Vector2i & imgSize) const
{
	return new RenderState(
		imgSize, scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max, MEMORYDEVICE_CUDA
	);
}

template<class TVoxel>
RenderState_VH* VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::CreateRenderState(const Scene<TVoxel, VoxelBlockHash> *scene, const Vector2i & imgSize) const
{
	return new RenderState_VH(
		VoxelBlockHash::noTotalEntries, imgSize, scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max, MEMORYDEVICE_CUDA
	);
}

template<class TVoxel>
void VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::FindVisibleBlocks(const Scene<TVoxel,VoxelBlockHash> *scene, const ORUtils::SE3Pose *pose, const Intrinsics *intrinsics, RenderState *renderState) const
{
	const HashEntry *hashTable = scene->index.GetEntries();
	int noTotalEntries = scene->index.noTotalEntries;
	float voxelSize = scene->sceneParams->voxelSize;
	Vector2i imgSize = renderState->renderingRangeImage->noDims;

	Matrix4f M = pose->GetM();
	Vector4f projParams = intrinsics->projectionParamsSimple.all;

	RenderState_VH *renderState_vh = (RenderState_VH*)renderState;

	ORcudaSafeCall(hipMemset(noVisibleEntries_device, 0, sizeof(int)));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));
	buildCompleteVisibleList_device << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, /*cacheStates, this->scene->useSwapping,*/ noTotalEntries,
		renderState_vh->GetVisibleEntryIDs(), noVisibleEntries_device, renderState_vh->GetEntriesVisibleType(), M, projParams, 
		imgSize, voxelSize);
	ORcudaKernelCheck;

	/*	if (this->scene->useSwapping)
			{
			reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries,
			noAllocatedVoxelEntries_device, entriesVisibleType);
			}*/

	ORcudaSafeCall(hipMemcpy(&renderState_vh->noVisibleEntries, noVisibleEntries_device, sizeof(int), hipMemcpyDeviceToHost));
}

template<class TVoxel, class TIndex>
int VisualisationEngine_CUDA<TVoxel, TIndex>::CountVisibleBlocks(const Scene<TVoxel,TIndex> *scene, const RenderState *renderState, int minBlockId, int maxBlockId) const
{
	return 1;
}

template<class TVoxel>
int VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::CountVisibleBlocks(const Scene<TVoxel,VoxelBlockHash> *scene, const RenderState *renderState, int minBlockId, int maxBlockId) const
{
	const RenderState_VH *renderState_vh = (const RenderState_VH*)renderState;

	int noVisibleEntries = renderState_vh->noVisibleEntries;
	const int *visibleEntryIDs_device = renderState_vh->GetVisibleEntryIDs();

	ORcudaSafeCall(hipMemset(noTotalBlocks_device, 0, sizeof(uint)));

	dim3 blockSize(256);
	dim3 gridSize((int)ceil((float)noVisibleEntries / (float)blockSize.x));

	const HashEntry *hashTable_device = scene->index.GetEntries();
	countVisibleBlocks_device<<<gridSize,blockSize>>>(visibleEntryIDs_device, noVisibleEntries, hashTable_device, noTotalBlocks_device, minBlockId, maxBlockId);
	ORcudaKernelCheck;

	uint noTotalBlocks;
	ORcudaSafeCall(hipMemcpy(&noTotalBlocks, noTotalBlocks_device, sizeof(uint), hipMemcpyDeviceToHost));

	return noTotalBlocks;
}

template<class TVoxel, class TIndex>
void VisualisationEngine_CUDA<TVoxel, TIndex>::CreateExpectedDepths(const Scene<TVoxel,TIndex> *scene, const ORUtils::SE3Pose *pose,
	const Intrinsics *intrinsics, RenderState *renderState) const
{
	Vector2f *minmaxData = renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA);

	Vector2f init;
	//TODO : this could be improved a bit...
	init.x = 0.2f; init.y = 3.0f;
	memsetKernel<Vector2f>(minmaxData, init, renderState->renderingRangeImage->dataSize);
}

template<class TVoxel>
void VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::CreateExpectedDepths(const Scene<TVoxel,VoxelBlockHash> *scene, const ORUtils::SE3Pose *pose, const Intrinsics *intrinsics,
	RenderState *renderState) const
{
	float voxelSize = scene->sceneParams->voxelSize;

	Vector2i imgSize = renderState->renderingRangeImage->noDims;
	Vector2f *minmaxData = renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA);

	Vector2f init;
	init.x = FAR_AWAY; init.y = VERY_CLOSE;
	memsetKernel<Vector2f>(minmaxData, init, renderState->renderingRangeImage->dataSize);

	RenderState_VH* renderState_vh = (RenderState_VH*)renderState;

	//go through list of visible 8x8x8 blocks
	{
		const HashEntry *hash_entries = scene->index.GetEntries();
		const int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
		int noVisibleEntries = renderState_vh->noVisibleEntries;
		if (noVisibleEntries == 0) return;

		dim3 blockSize(256);
		dim3 gridSize((int)ceil((float)noVisibleEntries / (float)blockSize.x));
		ORcudaSafeCall(hipMemset(noTotalBlocks_device, 0, sizeof(uint)));
		projectAndSplitBlocks_device << <gridSize, blockSize >> >(hash_entries, visibleEntryIDs, noVisibleEntries, pose->GetM(),
			intrinsics->projectionParamsSimple.all, imgSize, voxelSize, renderingBlockList_device, noTotalBlocks_device);
		ORcudaKernelCheck;
	}

	uint noTotalBlocks;
	ORcudaSafeCall(hipMemcpy(&noTotalBlocks, noTotalBlocks_device, sizeof(uint), hipMemcpyDeviceToHost));
	if (noTotalBlocks == 0) return;
	if (noTotalBlocks > (unsigned)MAX_RENDERING_BLOCKS) noTotalBlocks = MAX_RENDERING_BLOCKS;

	// go through rendering blocks
	{
		// fill minmaxData
		dim3 blockSize(16, 16);
		dim3 gridSize((unsigned int)ceil((float)noTotalBlocks / 4.0f), 4);
		fillBlocks_device << <gridSize, blockSize >> >(noTotalBlocks, renderingBlockList_device, imgSize, minmaxData);
		ORcudaKernelCheck;
	}
}

template <class TVoxel, class TIndex>
static void GenericRaycast(const Scene<TVoxel, TIndex> *scene, const Vector2i& imgSize, const Matrix4f& invM, const Vector4f& projParams, const RenderState *renderState, bool updateVisibleList)
{
	float voxelSize = scene->sceneParams->voxelSize;
	float oneOverVoxelSize = 1.0f / voxelSize;

	uchar *entriesVisibleType = NULL;
	if (updateVisibleList&&(dynamic_cast<const RenderState_VH*>(renderState)!=NULL))
	{
		entriesVisibleType = ((RenderState_VH*)renderState)->GetEntriesVisibleType();
	}

	dim3 cudaBlockSize(16, 12);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
	if (entriesVisibleType!=NULL) genericRaycast_device<TVoxel, VoxelBlockHash, true> << <gridSize, cudaBlockSize >> >(
			renderState->raycastResult->GetData(MEMORYDEVICE_CUDA),
			entriesVisibleType,
			scene->localVBA.GetVoxelBlocks(),
			scene->index.getIndexData(),
			imgSize,
			invM,
			InvertProjectionParams(projParams),
			oneOverVoxelSize,
			renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA),
			scene->sceneParams->mu
		);
	else genericRaycast_device<TVoxel, VoxelBlockHash, false> << <gridSize, cudaBlockSize >> >(
			renderState->raycastResult->GetData(MEMORYDEVICE_CUDA),
			NULL,
			scene->localVBA.GetVoxelBlocks(),
			scene->index.getIndexData(),
			imgSize,
			invM,
			InvertProjectionParams(projParams),
			oneOverVoxelSize,
			renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA),
			scene->sceneParams->mu
		);
	ORcudaKernelCheck;
}

template<class TVoxel, class TIndex>
static void RenderImage_common(const Scene<TVoxel, TIndex> *scene, const Matrix4f pose, const Vector4f intrinsics, const RenderState *renderState,
	UChar4Image *outputImage, IVisualisationEngine::RenderImageType type, IVisualisationEngine::RenderRaycastSelection raycastType)
{
	Vector2i imgSize = outputImage->noDims;
	// Matrix4f invM = pose->GetInvM();
	Matrix4f invM = pose.inv();

	Vector4f *pointsRay;
	if (raycastType == IVisualisationEngine::RENDER_FROM_OLD_RAYCAST) {
		pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);
	} else if (raycastType == IVisualisationEngine::RENDER_FROM_OLD_FORWARDPROJ) {
		pointsRay = renderState->forwardProjection->GetData(MEMORYDEVICE_CUDA);
	} else {
		GenericRaycast(scene, imgSize, invM, intrinsics, renderState, false);
		pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);
	}

	Vector3f lightSource = -Vector3f(invM.getColumn(2));

	Vector4u *outRendering = outputImage->GetData(MEMORYDEVICE_CUDA);

	dim3 cudaBlockSize(8, 8);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));

	if ((type == IVisualisationEngine::RENDER_COLOUR_FROM_VOLUME)&&
	    (!TVoxel::hasColorInformation)) type = IVisualisationEngine::RENDER_SHADED_GREYSCALE;

	switch (type) {
	case IVisualisationEngine::RENDER_COLOUR_FROM_VOLUME:
		renderColour_device<TVoxel, TIndex> <<<gridSize, cudaBlockSize>>>(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
			scene->index.getIndexData(), imgSize);
		ORcudaKernelCheck;
		break;
	case IVisualisationEngine::RENDER_COLOUR_FROM_NORMAL:
		renderColourFromNormal_device<TVoxel, TIndex> <<<gridSize, cudaBlockSize>>>(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
			scene->index.getIndexData(), imgSize, lightSource);
		ORcudaKernelCheck;
		break;
	case IVisualisationEngine::RENDER_COLOUR_FROM_CONFIDENCE:
		renderColourFromConfidence_device<TVoxel, TIndex> <<<gridSize, cudaBlockSize>>>(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
			scene->index.getIndexData(), imgSize, lightSource);
		ORcudaKernelCheck;
		break;
	case IVisualisationEngine::RENDER_SHADED_GREYSCALE_IMAGENORMALS:
		if (intrinsics->FocalLengthSignsDiffer())
		{
			renderGrey_ImageNormals_device<true> <<<gridSize, cudaBlockSize>>>(outRendering, pointsRay, scene->sceneParams->voxelSize,
				imgSize, lightSource);
		}
		else
		{
			renderGrey_ImageNormals_device<false> <<<gridSize, cudaBlockSize>>>(outRendering, pointsRay, scene->sceneParams->voxelSize,
				imgSize, lightSource);
		}
		ORcudaKernelCheck;
		break;
	case IVisualisationEngine::RENDER_SHADED_GREYSCALE:
	default:
		renderGrey_device<TVoxel, TIndex> <<<gridSize, cudaBlockSize>>>(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
			scene->index.getIndexData(), imgSize, lightSource);
		ORcudaKernelCheck;
		break;
	}
}

template<class TVoxel, class TIndex>
static void CreatePointCloud_common(const Scene<TVoxel, TIndex> *scene, const View *view, TrackingState *trackingState, RenderState *renderState,
	bool skipPoints, uint *noTotalPoints_device)
{
	Vector2i imgSize = renderState->raycastResult->noDims;
	Matrix4f invM = trackingState->pose_d->GetInvM() * view->calib.trafo_rgb_to_depth.calib;

	GenericRaycast(scene, imgSize, invM, view->calib.intrinsics_rgb.projectionParamsSimple.all, renderState, true);
	trackingState->pose_pointCloud->SetFrom(trackingState->pose_d);

	ORcudaSafeCall(hipMemsetAsync(noTotalPoints_device, 0, sizeof(uint)));

	Vector3f lightSource = -Vector3f(invM.getColumn(2));
	Vector4f *locations = trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
	Vector4f *colours = trackingState->pointCloud->colours->GetData(MEMORYDEVICE_CUDA);
	Vector4f *pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);

	dim3 cudaBlockSize(16, 16);
	dim3 gridSize = getGridSize(imgSize, cudaBlockSize);
	renderPointCloud_device<TVoxel, TIndex> << <gridSize, cudaBlockSize >> >(locations, colours, noTotalPoints_device,
		pointsRay, scene->localVBA.GetVoxelBlocks(), scene->index.getIndexData(), skipPoints, scene->sceneParams->voxelSize, imgSize, lightSource);
	ORcudaKernelCheck;

	ORcudaSafeCall(hipMemcpy(&trackingState->pointCloud->noTotalPoints, noTotalPoints_device, sizeof(uint), hipMemcpyDeviceToHost));
}

template<class TVoxel, class TIndex>
void CreateICPMaps_common(const Scene<TVoxel, TIndex> *scene, const View *view, TrackingState *trackingState, RenderState *renderState)
{
	Vector2i imgSize = renderState->raycastResult->noDims;
	Matrix4f invM = trackingState->pose_d->GetInvM();

	GenericRaycast(scene, imgSize, invM, view->calib.intrinsics_d.projectionParamsSimple.all, renderState, true);
	trackingState->pose_pointCloud->SetFrom(trackingState->pose_d);

	Vector4f *pointsMap = trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
	Vector4f *normalsMap = trackingState->pointCloud->colours->GetData(MEMORYDEVICE_CUDA);
	Vector4f *pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);
	Vector3f lightSource = -Vector3f(invM.getColumn(2));

	dim3 cudaBlockSize(16, 12);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));

	if (view->calib.intrinsics_d.FocalLengthSignsDiffer())
	{
		renderICP_device<true> <<<gridSize, cudaBlockSize>>>(pointsMap, normalsMap, pointsRay,
			scene->sceneParams->voxelSize, imgSize, lightSource);
	}
	else
	{
		renderICP_device<false> <<<gridSize, cudaBlockSize>>>(pointsMap, normalsMap, pointsRay,
			scene->sceneParams->voxelSize, imgSize, lightSource);
	}
	ORcudaKernelCheck;
}

template<class TVoxel, class TIndex>
static void ForwardRender_common(const Scene<TVoxel, TIndex> *scene, const View *view, TrackingState *trackingState, RenderState *renderState, 
	uint *noTotalPoints_device)
{
	Vector2i imgSize = renderState->raycastResult->noDims;
	Matrix4f M = trackingState->pose_d->GetM();
	Matrix4f invM = trackingState->pose_d->GetInvM();
	const Vector4f& projParams = view->calib.intrinsics_d.projectionParamsSimple.all;

	const Vector4f *pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);
	float *currentDepth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4f *forwardProjection = renderState->forwardProjection->GetData(MEMORYDEVICE_CUDA);
	int *fwdProjMissingPoints = renderState->fwdProjMissingPoints->GetData(MEMORYDEVICE_CUDA);
	const Vector2f *minmaximg = renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA);
	float oneOverVoxelSize = 1.0f / scene->sceneParams->voxelSize;
	float voxelSize = scene->sceneParams->voxelSize;
	const TVoxel *voxelData = scene->localVBA.GetVoxelBlocks();
	const typename TIndex::IndexData *voxelIndex = scene->index.getIndexData();

	renderState->forwardProjection->Clear();

	dim3 blockSize, gridSize;

	{ // forward projection
		blockSize = dim3(16, 16);
		gridSize = dim3((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

		forwardProject_device << <gridSize, blockSize >> >(forwardProjection, pointsRay, imgSize, M, projParams, voxelSize);
		ORcudaKernelCheck;
	}

	ORcudaSafeCall(hipMemset(noTotalPoints_device, 0, sizeof(uint)));

	{ // find missing points
		blockSize = dim3(16, 16);
		gridSize = dim3((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

		findMissingPoints_device << <gridSize, blockSize >> >(fwdProjMissingPoints, noTotalPoints_device, minmaximg, 
			forwardProjection, currentDepth, imgSize);
		ORcudaKernelCheck;
	}

	ORcudaSafeCall(hipMemcpy(&renderState->noFwdProjMissingPoints, noTotalPoints_device, sizeof(uint), hipMemcpyDeviceToHost));

	{ // render missing points
		blockSize = dim3(256);
		gridSize = dim3((int)ceil((float)renderState->noFwdProjMissingPoints / blockSize.x));

		genericRaycastMissingPoints_device<TVoxel, TIndex, false> << <gridSize, blockSize >> >(forwardProjection, NULL, voxelData, voxelIndex, imgSize, invM,
			InvertProjectionParams(projParams), oneOverVoxelSize, fwdProjMissingPoints, renderState->noFwdProjMissingPoints, minmaximg, scene->sceneParams->mu);
		ORcudaKernelCheck;
	}
}

// template<class TVoxel, class TIndex>
// void VisualisationEngine_CUDA<TVoxel, TIndex>::RenderImage(const Scene<TVoxel,TIndex> *scene, const ORUtils::SE3Pose *pose, const Intrinsics *intrinsics, const RenderState *renderState,
// 	UChar4Image *outputImage, IVisualisationEngine::RenderImageType type,
// 	IVisualisationEngine::RenderRaycastSelection raycastType) const
// {
// 	RenderImage_common(scene, pose, intrinsics, renderState, outputImage, type, raycastType);
// }
template<class TVoxel, class TIndex>
void VisualisationEngine_CUDA<TVoxel, TIndex>::RenderImage(const Scene<TVoxel,TIndex> *scene, Matrix4f pose, const Vector4f intrinsics, const RenderState *renderState,
	UChar4Image *outputImage, IVisualisationEngine::RenderImageType type,
	IVisualisationEngine::RenderRaycastSelection raycastType) const
{
	RenderImage_common(scene, pose, intrinsics, renderState, outputImage, type, raycastType);
}

// template<class TVoxel>
// void VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::RenderImage(const Scene<TVoxel,VoxelBlockHash> *scene, const ORUtils::SE3Pose *pose, const Intrinsics *intrinsics,
// 	const RenderState *renderState, UChar4Image *outputImage, IVisualisationEngine::RenderImageType type,
// 	IVisualisationEngine::RenderRaycastSelection raycastType) const
// {
// 	RenderImage_common(scene, pose, intrinsics, renderState, outputImage, type, raycastType);
// }

template<class TVoxel, class TIndex>
void VisualisationEngine_CUDA<TVoxel, TIndex>::FindSurface(const Scene<TVoxel,TIndex> *scene, const ORUtils::SE3Pose *pose, const Intrinsics *intrinsics, const RenderState *renderState) const
{
	GenericRaycast(scene, renderState->raycastResult->noDims, pose->GetInvM(), intrinsics->projectionParamsSimple.all, renderState, false);
}

template<class TVoxel>
void VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::FindSurface(const Scene<TVoxel,VoxelBlockHash> *scene, const ORUtils::SE3Pose *pose, const Intrinsics *intrinsics,
	const RenderState *renderState) const
{
	GenericRaycast(scene, renderState->raycastResult->noDims, pose->GetInvM(), intrinsics->projectionParamsSimple.all, renderState, false);
}

template<class TVoxel, class TIndex>
void VisualisationEngine_CUDA<TVoxel, TIndex>::CreatePointCloud(const Scene<TVoxel,TIndex> *scene, const View *view, TrackingState *trackingState, RenderState *renderState, 
	bool skipPoints) const
{
	CreatePointCloud_common(scene, view, trackingState, renderState, skipPoints, noTotalPoints_device);
}

template<class TVoxel>
void VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::CreatePointCloud(const Scene<TVoxel,VoxelBlockHash> *scene,const View *view, TrackingState *trackingState, 
	RenderState *renderState, bool skipPoints) const
{
	CreatePointCloud_common(scene, view, trackingState, renderState, skipPoints, noTotalPoints_device);
}

template<class TVoxel, class TIndex>
void VisualisationEngine_CUDA<TVoxel, TIndex>::CreateICPMaps(const Scene<TVoxel,TIndex> *scene, const View *view, TrackingState *trackingState, 
	RenderState *renderState) const
{
	CreateICPMaps_common(scene, view, trackingState, renderState);
}

template<class TVoxel>
void VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::CreateICPMaps(const Scene<TVoxel,VoxelBlockHash> *scene, const View *view, TrackingState *trackingState, 
	RenderState *renderState) const
{
	CreateICPMaps_common(scene, view, trackingState, renderState);
}

template<class TVoxel, class TIndex>
void VisualisationEngine_CUDA<TVoxel, TIndex>::ForwardRender(const Scene<TVoxel,TIndex> *scene, const View *view, TrackingState *trackingState, 
	RenderState *renderState) const
{
	ForwardRender_common(scene, view, trackingState, renderState, this->noTotalPoints_device);
}

template<class TVoxel>
void VisualisationEngine_CUDA<TVoxel, VoxelBlockHash>::ForwardRender(const Scene<TVoxel,VoxelBlockHash> *scene, const View *view, TrackingState *trackingState, 
	RenderState *renderState) const
{
	ForwardRender_common(scene, view, trackingState, renderState, this->noTotalPoints_device);
}