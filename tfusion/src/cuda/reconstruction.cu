#include "hip/hip_runtime.h"
#include <tfusion/cuda/reconstruction_CUDA.hpp>

struct AllocationTempData{

	int noAllocatedVoxelEntries;
	int noAllocatedExcessEntries;
	int noVisibleEntries; 
};

using namespace tfusion;

namespace{
	__global__ void integrateIntoScene_device(TVoxel *localVBA, const HashEntry *hashTable, int *noVisibleEntryIDs,
		const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, const float *confidence, Vector2i imgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
		Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

	template<class TVoxel, bool stopMaxW>
	__global__ void integrateIntoScene_device(TVoxel *voxelArray, const PlainVoxelArray::VoxelArrayInfo *arrayInfo,
		const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, const float *confidence, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
		Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

	__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
		Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, HashEntry *hashTable, float viewFrustum_min,
		float viewFrustrum_max);

	__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, HashEntry *hashTable, int noTotalEntries,
		AllocationTempData *allocData, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords);

	__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, HashEntry *hashTable, int noTotalEntries,
		AllocationTempData *allocData, uchar *entriesVisibleType);

	__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries);

	template<bool useSwapping>
	__global__ void buildVisibleList_device(HashEntry *hashTable, HashSwapState *swapStates, int noTotalEntries,
		int *visibleEntryIDs, AllocationTempData *allocData, uchar *entriesVisibleType,
		Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize);
}

//host method

template<class TVoxel>
SceneReconstruction<TVoxel,VoxelBlockHash>::SceneReconstruction(void)
{
	ORcudaSafeCall(hipMalloc((void**)&allocationTempData_device, sizeof(AllocationTempData)));
	ORcudaSafeCall(hipHostMalloc((void**)&allocationTempData_host, sizeof(AllocationTempData)));

	int noTotalEntries = VoxelBlockHash::noTotalEntries;
	ORcudaSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ORcudaSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector4s)));
}

template<class TVoxel>
SceneReconstruction<TVoxel,VoxelBlockHash>::~SceneReconstruction(void)
{
	ORcudaSafeCall(hipHostFree(allocationTempData_host));
	ORcudaSafeCall(hipFree(allocationTempData_device));
	ORcudaSafeCall(hipFree(entriesAllocType_device));
	ORcudaSafeCall(hipFree(blockCoords_device));
}

template<class TVoxel>
void SceneReconstruction<TVoxel,VoxelBlockHash>::ResetScene(Scene<TVoxel, VoxelBlockHash> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;

	HashEntry tmpEntry;
	memset(&tmpEntry, 0, sizeof(HashEntry));
	tmpEntry.ptr = -2;
	HashEntry *hashEntry_ptr = scene->index.GetEntries();
	memsetKernel<HashEntry>(hashEntry_ptr, tmpEntry, scene->index.noTotalEntries);
	int *excessList_ptr = scene->index.GetExcessAllocationList();
	fillArrayKernel<int>(excessList_ptr, SDF_EXCESS_LIST_SIZE);

	scene->index.SetLastFreeExcessListId(SDF_EXCESS_LIST_SIZE - 1);
}
//modified by chuan
template<class TVoxel>
void SceneReconstruction<TVoxel, VoxelBlockHash>::AllocateSceneFromDepth(Scene<TVoxel, VoxelBlockHash> *scene, const Intr intr, 
	const Matrix4f pose, cuda::Dist &dist,const RenderState *renderState,bool onlyUpdateVisibleList, bool resetVisibleList)
{
	// Vector2i depthImgSize = view->depth->noDims;
	Vector2i depthImgSize(dist.cols,dist.rows);
	float voxelSize = scene->sceneParams->voxelSize;

	// Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	RenderState_VH *renderState_vh = (RenderState_VH*)renderState;

	if (resetVisibleList) renderState_vh->noVisibleEntries = 0;

	// Matrix4f M_d(pose.matrix(0,0),pose.matrix(0,1),pose.matrix(0,2),pose.matrix(0,3),
	// 			pose.matrix(1,0),pose.matrix(1,1),pose.matrix(1,2),pose.matrix(1,3),
	// 			pose.matrix(2,0),pose.matrix(2,1),pose.matrix(2,2),pose.matrix(2,3),
	// 			pose.matrix(3,0),pose.matrix(3,1),pose.matrix(3,2),pose.matrix(3,3));
	// M_d = trackingState->pose_d->GetM(); M_d.inv(invM_d);
	Matrix4f M_d(pose);
	Matrix4f invM_d;
	M_d.inv(invM_d);
	
	projParams_d = new Vector4f(intr.fx,intr.fy,intr.cx,intr.cy);
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	// float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	// ushort *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	HashEntry *hashTable = scene->index.GetEntries();
	HashSwapState *swapStates = scene->globalCache != NULL ? scene->globalCache->GetSwapStates(true) : 0;

	int noTotalEntries = scene->index.noTotalEntries;

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeVS(256, 1);
	dim3 gridSizeVS((int)ceil((float)renderState_vh->noVisibleEntries / (float)cudaBlockSizeVS.x));

	float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);

	AllocationTempData *tempData = (AllocationTempData*)allocationTempData_host;
	tempData->noAllocatedVoxelEntries = scene->localVBA.lastFreeBlockId;
	tempData->noAllocatedExcessEntries = scene->index.GetLastFreeExcessListId();
	tempData->noVisibleEntries = 0;
	ORcudaSafeCall(hipMemcpyAsync(allocationTempData_device, tempData, sizeof(AllocationTempData), hipMemcpyHostToDevice));

	ORcudaSafeCall(hipMemsetAsync(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));

	if (gridSizeVS.x > 0)
	{
		setToType3 << <gridSizeVS, cudaBlockSizeVS >> > (entriesVisibleType, visibleEntryIDs, renderState_vh->noVisibleEntries);
		ORcudaKernelCheck;
	}

	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType, 
		blockCoords_device, dist, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);
	ORcudaKernelCheck;

	bool useSwapping = scene->globalCache != NULL;
	if (onlyUpdateVisibleList) useSwapping = false;
	//execute
	if (!onlyUpdateVisibleList)
	{
		allocateVoxelBlocksList_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, excessAllocationList, hashTable,
			noTotalEntries, (AllocationTempData*)allocationTempData_device, entriesAllocType_device, entriesVisibleType,
			blockCoords_device);
		ORcudaKernelCheck;
	}
	//no execute
	if (useSwapping)
	{
		buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize);
		ORcudaKernelCheck;
	}
	//execute
	else
	{
		buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize);
		ORcudaKernelCheck;
	}
	//no execute
	if (useSwapping)
	{
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries, 
			(AllocationTempData*)allocationTempData_device, entriesVisibleType);
		ORcudaKernelCheck;
	}

	ORcudaSafeCall(hipMemcpy(tempData, allocationTempData_device, sizeof(AllocationTempData), hipMemcpyDeviceToHost));
	renderState_vh->noVisibleEntries = tempData->noVisibleEntries;
	scene->localVBA.lastFreeBlockId = tempData->noAllocatedVoxelEntries;
	scene->index.SetLastFreeExcessListId(tempData->noAllocatedExcessEntries);
}

template<class TVoxel>
void SceneReconstruction<TVoxel, VoxelBlockHash>::IntegrateIntoScene(Scene<TVoxel, VoxelBlockHash> *scene, const Intr intr,
	const Matrix4f pose, cuda::Dist& dist, const RenderState *renderState)
{
	Vector2i depthImgSize(dist.cols,dist.rows);
	float voxelSize = scene->sceneParams->voxelSize;

	RenderState_VH *renderState_vh = (RenderState_VH*)renderState;
	if(renderState_vh->noVisibleEntries == 0) return;

	// M_d = trackingState->pose_d->GetM();
	// Matrix4f M_d(pose.matrix(0,0),pose.matrix(0,1),pose.matrix(0,2),pose.matrix(0,3),
	// 			pose.matrix(1,0),pose.matrix(1,1),pose.matrix(1,2),pose.matrix(1,3),
	// 			pose.matrix(2,0),pose.matrix(2,1),pose.matrix(2,2),pose.matrix(2,3),
	// 			pose.matrix(3,0),pose.matrix(3,1),pose.matrix(3,2),pose.matrix(3,3));
	Matrix4f M_d(pose);
	// if (TVoxel::hasColorInformation) M_rgb = view->calib.trafo_rgb_to_depth.calib_inv * M_d;

	Vector4f projParams_d(intr.fx,intr.fy,intr.cx,intr.cy);

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	// float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	// float *confidence = view->depthConfidence->GetData(MEMORYDEVICE_CUDA);
	// Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	HashEntry *hashTable = scene->index.GetEntries();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noVisibleEntries);

	if (scene->sceneParams->stopIntegratingAtMaxW)
	{
		// integrateIntoScene_device<TVoxel, true> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
		// 	rgb, rgbImgSize, depth, confidence, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		// ORcudaKernelCheck;
		integrateIntoScene_device<TVoxel, true> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,(PtrStepSz<ushort>)dist, depthImgSize, M_d, projParams_d, voxelSize, mu, maxW);
		ORcudaKernelCheck;
	}
	//execute
	else
	{
		// integrateIntoScene_device<TVoxel, false> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
		// 	rgb, rgbImgSize, depth, confidence, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		// ORcudaKernelCheck;
		integrateIntoScene_device<TVoxel, false> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,(PtrStepSz<ushort>)dist, depthImgSize, M_d, projParams_d, voxelSize, mu, maxW);
		ORcudaKernelCheck;
	}
}

namespace
{
	//device functions

template<class TVoxel, bool stopMaxW>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const HashEntry *hashTable, int *visibleEntryIDs,
	const PtrStepSz<ushort> depth, Vector2i depthImgSize, Matrix4f M_d, Vector4f projParams_d, 
	float _voxelSize, float mu, int maxW)
{
	Vector3i globalPos;
	int entryId = visibleEntryIDs[blockIdx.x];

	const HashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr < 0) return;

	globalPos = currentHashEntry.pos.toInt() * SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (stopMaxW) if (localVoxelBlock[locId].w_depth == maxW) return;
	//if (approximateIntegration) if (localVoxelBlock[locId].w_depth != 0) return;

	pt_model.x = (float)(globalPos.x + x) * _voxelSize;
	pt_model.y = (float)(globalPos.y + y) * _voxelSize;
	pt_model.z = (float)(globalPos.z + z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation, TVoxel::hasConfidenceInformation, TVoxel>::compute(localVoxelBlock[locId], 
		pt_model, M_d, projParams_d, mu, maxW, depth, depthImgSize);
}

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const cuda::Dist &depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, HashEntry *hashTable, float viewFrustum_min,
	float viewFrustum_max)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, (tfusion::cuda::PtrStepSz<ushort>)depth, invM_d,
		projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noVisibleEntries - 1) return;
	entriesVisibleType[visibleEntryIDs[entryId]] = 3;
}

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, HashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;

	switch (entriesAllocType[targetIdx])
	{
	case 1: //needs allocation, fits in the ordered list
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			HashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;

			hashTable[targetIdx] = hashEntry;
		}
		else
		{
			// Mark entry as not visible since we couldn't allocate it but buildHashAllocAndVisibleTypePP changed its state.
			entriesVisibleType[targetIdx] = 0;

			// Restore the previous value to avoid leaks.
			atomicAdd(&allocData->noAllocatedVoxelEntries, 1);
		}
		break;

	case 2: //needs allocation in the excess list
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		exlIdx = atomicSub(&allocData->noAllocatedExcessEntries, 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			HashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;

			int exlOffset = excessAllocationList[exlIdx];

			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			hashTable[SDF_BUCKET_NUM + exlOffset] = hashEntry; //add child to the excess list

			entriesVisibleType[SDF_BUCKET_NUM + exlOffset] = 1; //make child visible
		}
		else
		{
			// No need to mark the entry as not visible since buildHashAllocAndVisibleTypePP did not mark it.
			// Restore the previous values to avoid leaks.
			atomicAdd(&allocData->noAllocatedVoxelEntries, 1);
			atomicAdd(&allocData->noAllocatedExcessEntries, 1);
		}

		break;
	}
}

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, HashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, /*int *noAllocatedVoxelEntries,*/ uchar *entriesVisibleType)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx;
	int hashEntry_ptr = hashTable[targetIdx].ptr;

	if (entriesVisibleType[targetIdx] > 0 && hashEntry_ptr == -1) //it is visible and has been previously allocated inside the hash, but deallocated from VBA
	{
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		if (vbaIdx >= 0) hashTable[targetIdx].ptr = voxelAllocationList[vbaIdx];
		else atomicAdd(&allocData->noAllocatedVoxelEntries, 1);
	}
}

template<bool useSwapping>
__global__ void buildVisibleList_device(HashEntry *hashTable, HashSwapState *swapStates, int noTotalEntries,
	int *visibleEntryIDs, AllocationTempData *allocData, uchar *entriesVisibleType, 
	Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;
	shouldPrefix = false;
	__syncthreads();

	unsigned char hashVisibleType = entriesVisibleType[targetIdx];
	const HashEntry & hashEntry = hashTable[targetIdx];

	if (hashVisibleType == 3)
	{
		bool isVisibleEnlarged, isVisible;

		if (useSwapping)
		{
			checkBlockVisibility<true>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, depthImgSize);
			if (!isVisibleEnlarged) hashVisibleType = 0;
		} else {
			checkBlockVisibility<false>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, depthImgSize);
			if (!isVisible) hashVisibleType = 0;
		}
		entriesVisibleType[targetIdx] = hashVisibleType;
	}

	if (hashVisibleType > 0) shouldPrefix = true;

	if (useSwapping)
	{
		if (hashVisibleType > 0 && swapStates[targetIdx].state != 2) swapStates[targetIdx].state = 1;
	}

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0, &allocData->noVisibleEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) visibleEntryIDs[offset] = targetIdx;
	}

}

}
